#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for SAXPY: y[i] = a * x[i] + y[i]
__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

int main() {
    int N = 1000000;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *x, *y;
    x = (float *)malloc(size);
    y = (float *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Allocate device memory
    float *d_x, *d_y;
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    // Copy host data to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Kernel launch configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch SAXPY kernel
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(N, 2.0f, d_x, d_y);

    // Copy result back to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Print first result
    printf("y[0] = %f\n", y[0]);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    // Free host memory
    free(x);
    free(y);

    return 0;
}
