#include "hip/hip_runtime.h"
// Filename: mm_coalesced.cu
// Coalesced kernel : swapped x/y mapping improves memory access
template <typename T>
__global__ void mm_coalesced_kernel(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n, size_t p)
{
    size_t j { blockIdx.x * blockDim.x + threadIdx.x }; // columns -> x
    size_t i { blockIdx.y * blockDim.y + threadIdx.y }; // rows -> y
    if ((i >= m) || (j >= p)) return;
    // Threads now traverse rows/cols in contiguous order
    T acc_sum { 0 };
    for (size_t k { 0 }; k < n; ++k)
        acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
    mat_3[i * p + j] = acc_sum;
}

// Main Function
// Same structure as Problem 1
int main()
{
    const size_t num_tests { 2 };
    assert(random_multiple_test_mm_cuda<int32_t>(num_tests));
    assert(random_multiple_test_mm_cuda<float>(num_tests));
    assert(random_multiple_test_mm_cuda<double>(num_tests));
    std::cout << "All tests passed!\n";
    const size_t num_measurement_tests { 2 };
    const size_t num_measurement_warmups { 1 };
    size_t m { MAT_DIM }, n { MAT_DIM }, p { MAT_DIM };
    float mm_cuda_int32_latency = measure_latency_mm_cuda<int32_t>(m, n, p, num_measurement_tests, num_measurement_warmups);
    float mm_cuda_float_latency = measure_latency_mm_cuda<float>(m, n, p, num_measurement_tests, num_measurement_warmups);
    float mm_cuda_double_latency = measure_latency_mm_cuda<double>(m, n, p, num_measurement_tests, num_measurement_warmups);
    std::cout << "Matrix Multiplication Runtime\n";
    std::cout << "m: " << m << " n: " << n << " p: " << p << "\n";
    std::cout << "INT32: " << mm_cuda_int32_latency << " ms\n";
    std::cout << "FLOAT: " << mm_cuda_float_latency << " ms\n";
    std::cout << "DOUBLE: " << mm_cuda_double_latency << " ms\n";
    return 0;
}
